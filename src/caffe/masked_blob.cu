#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/blob.hpp"
#include "caffe/masked_blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {
template <typename Dtype>
__global__ void ApplyMask(const int n, const Dtype* in_diff,
    const Dtype* mask, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * mask[index];
    //out_diff[index] = mask[index]?in_diff[index]:(Dtype)0;
  }
}

template <typename Dtype>
void MaskedBlob<Dtype>::_apply_mask_gpu() {
  Dtype* weights_diff = static_cast<Dtype*>(this->diff_->mutable_gpu_data());
  const int count = this->count();
  const Dtype* mask = static_cast<const Dtype*>(this->mask_->gpu_data());
  ApplyMask<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(count,weights_diff,mask,weights_diff);
}

template void MaskedBlob<float>::_apply_mask_gpu();
template void MaskedBlob<double>::_apply_mask_gpu();

}